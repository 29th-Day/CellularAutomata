#include "hip/hip_runtime.h"
#include "activations.h"

#include <cmath>

#include <hip/hip_runtime.h>

#define EULER_NUMBER_F 2.71828182846f

template <typename T>
__device__ __host__ T Activations::_normal(T x)
{
    if (x < 0)
        return static_cast<T>(0);
    else if (x > 1)
        return static_cast<T>(1);
    else
        return x;
}

template __device__ __host__ float Activations::_normal(float);

template <typename T>
__device__ __host__ T Activations::_life(T x)
{
    unsigned char u = static_cast<unsigned char>(x);

    unsigned char neighbours = u & 0xF; // low  nibble
    bool alive = u & 0xF0;              // high nibble

    switch (neighbours)
    {
    case 2:
        // staying alive
        return static_cast<T>((alive) ? 1 : 0);
    case 3:
        // birth
        return static_cast<T>(1);
    default:
        // under- / overpopulation
        return static_cast<T>(0);
    }
}

template __device__ __host__ float Activations::_life(float);

// template <typename T>
// device host T Activations::sigmoid(T x)
// {
//     return (1 / (1 + std::pow(EULER_NUMBER_F, -x)));
// }

// template <typename T>
// device host T Activations::sin(T x)
// {
//     return std::sin(x);
// }

// template <typename T>
// device host T Activations::cos(T x)
// {
//     return std::cos(x);
// }

// template <typename T>
// device host T Activations::tan(T x)
// {
//     return std::tan(x);
// }

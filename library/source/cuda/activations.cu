#include "hip/hip_runtime.h"
#include "activations.h"

#include <math.h>

#define EULER_NUMBER_F 2.71828182846f

__device__ float Activations::sigmoid(float x)
{
    return (1 / (1 + powf(EULER_NUMBER_F, -x)));
}

__device__ float Activations::life(float x)
{
    int neighbors = (int)x;
    bool alive = (x - neighbors) > 0.1;

    switch (neighbors)
    {
    case 2:
        // staying alive
        return (alive) ? 1.0f : 0.0f;
    case 3:
        // birth
        return 1.0f;
    default:
        // under- / overpopulation
        return 0.0f;
    }
}

__device__ float Activations::clip(float x)
{
    if (x < 0.0f)
        return 0.0f;
    else if (x > 1.0f)
        return 1.0f;
    else
        return x;
}

__device__ float Activations::sin(float x)
{
    return sinf(x);
}

__device__ float Activations::cos(float x)
{
    return cosf(x);
}

__device__ float Activations::tan(float x)
{
    return tanf(x);
}
